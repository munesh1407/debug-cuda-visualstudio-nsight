#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""

#include <GL/glut.h>
using namespace std;

double MinRe = -2.0;
double MaxRe = 1.0;
double MinIm = -1.2;

__global__ void mandelbrotKernel(double MaxIm, double Im_factor,double Re_factor,unsigned MaxIterations) {
	int y = threadIdx.x + blockIdx.x*blockDim.x;
	int x = threadIdx.y + blockIdx.y*blockDim.y;
	double c_im = MaxIm - y*Im_factor;
	double c_re = MinRe + x*Re_factor;

	double Z_re = c_re, Z_im = c_im;
	bool isInside = true;
	for (unsigned n = 0; n<MaxIterations; ++n)
	{
		double Z_re2 = Z_re*Z_re, Z_im2 = Z_im*Z_im;
		if (Z_re2 + Z_im2 > 4)
		{
			isInside = false;
			break;
		}
		Z_im = 2 * Z_re*Z_im + c_im;
		Z_re = Z_re2 - Z_im2 + c_re;
	}
	if (isInside)
	{
		glBegin(GL_POINTS);
			glColor3f(0, 0, 0);
			glVertex2i(x, y);
		glEnd();
	}

}

void drawMandelbrot()
{
	unsigned ImageHeight = glutGet(GLUT_WINDOW_HEIGHT);
	unsigned ImageWidth = glutGet(GLUT_WINDOW_WIDTH);
	double MaxIm = MinIm + (MaxRe - MinRe)*ImageHeight / ImageWidth;
	double Re_factor = (MaxRe - MinRe) / (ImageWidth - 1);
	double Im_factor = (MaxIm - MinIm) / (ImageHeight - 1);
	unsigned MaxIterations = 30;

	//glBegin(GL_POINTS);
	
	dim3 dimBlock(32, 32);
	unsigned blockRows = (ImageHeight % 32 ? ImageHeight / 32 : ImageHeight / 32 + 1);
	unsigned blockCols = (ImageWidth % 32 ? ImageWidth / 32 : ImageWidth / 32 + 1);
	dim3 dimGrid(blockRows, blockCols);
	mandelbrotKernel <<<dimGrid,dimBlock>>>(MaxIm,Im_factor,Re_factor,MaxIterations);
	
	//glEnd();
}

void display()
{
	glClear(GL_COLOR_BUFFER_BIT);

	drawMandelbrot();
	glFlush();
}

void init()
{
	glClearColor(1, 1, 1, 1);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0, glutGet(GLUT_WINDOW_WIDTH), 0, glutGet(GLUT_WINDOW_HEIGHT));
}

int main(int argc, char** argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB);
	glutInitWindowPosition(50, 50);
	glutInitWindowSize(600, 600);
	glutCreateWindow("Mandelbrot Set");

	init();
	glutDisplayFunc(display);
	glutMainLoop();
	return 0;
}
